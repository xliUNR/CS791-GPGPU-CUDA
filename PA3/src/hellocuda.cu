///////////////////////////////////////////////////////////////////////////////
//////////////////// kNN implementation main file /////////////////////////////
///////////////////// Written by Eric Li //////////////////////////////////////

//Includes
#include <cstdio>
#include <iostream>
#include <time.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "knn.h"
//define error macro
#define HANDLE_ERROR(func) { GPUAssert((func), __FILE__, __LINE__);}
inline void GPUAssert( hipError_t errCode, const char *file, int line, bool abort=true)
    {
     if( errCode != hipSuccess )
         {
          fprintf(stderr, "GPUAssert: %s %s %d\n", hipGetErrorString(errCode), file, line);
          if (abort) exit(errCode);
         }
    }

//main function
int main(int argc, char const *argv[])
{
   //initialize variables
   FILE * fp;
   int rows, cols;
   float *inData, *partial, *sortArray; 
   char* buffer;
   char* buffer2;
   size_t len;
   char* str;
   //ask user for dimension of input data matrix
   std::cout << " Please enter amount of rows desired to read in: ";
   std::cin >> rows;
   
   std::cout << " Please enter amount of columns desired to read in: ";
   std::cin >> cols;

   //allocate Unified memory for input data storage
   HANDLE_ERROR( hipMallocManaged( &inData, rows*cols*sizeof(float)) );
   HANDLE_ERROR( hipMallocManaged( &partial, rows*cols*sizeof(float)) );
   HANDLE_ERROR( hipMallocManaged( &sortArray, rows*sizeof(float)) );
   
   //allocate memory for read buffer
   buffer = (char*) malloc(cols*sizeof(float));
   buffer2 = (char*) malloc(20*sizeof(float));
   //open file and read in data
   fp = fopen("../src/PA3_nrdc_data.csv", "r");
   
   //test for successful file opening
   if(fp){
      /*for(int i = 0; i < rows; i++){
         fgets(buffer, rows*sizeof(float), fp);

         for(int j = 0; j < cols; j++){

         }
      }*/
     //std::fin.ignore(' '); 
     getdelim(&buffer2, &len, ' ,', fp);
     fgets(buffer, cols*sizeof(double), fp);
     std::cout << std::endl << "This is the string printed: " << buffer;
     str = strtok(buffer, " ,");
     std::cout << std::endl << "This is the string printed: " << str ;
   }

   else{
      std::cout << std::endl << "File opening error, please try again";
   }
   //read in data from file
   fclose(fp);



   //free memory
   hipFree(inData);
   hipFree(partial);
   hipFree(sortArray);
   free(buffer);
   free(buffer2);
   return 0;
}