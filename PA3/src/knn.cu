#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
////////////// Device code for PA3: kNN data Imputation    ////////////////////
//////////////////////////  written by Eric Li ////////////////////////////////
////////////////////////////////////////////////////////////////////////////// 

#include "knn.h"

/*
  This is the main function that performs the kNN algorithm.
*/

__global__ void knnDist( float *inputMat, float *partialMat, int imputRow, 
                                                           int rows, int cols){
   //initialize variables
   int bidx, tidx, reduceThreads, sumIdx, EmptyoffsetIndex, imputIdx;
   float diff;
   /*
     calculate unique index in matrix. This is so that each thread can access
     the correct memory location corresponding to it's data point. This index 
     is calculated from the size of the array and the block index and thread 
     indices of each thread.
   */  
   bidx = blockIdx.x;
   //reduceThreads = cols / 2;

   while( bidx < rows )
      {  
         /*
           calc thread index in partial matrix, offset by 2 since first col is
           id and the second col contains holes 
         */
         tidx = bidx * cols + threadIdx.x + 2;

         //Calculate offset of 2nd col, which tells whether row has hole or not
         EmptyoffsetIndex = ( bidx * cols + 1 );
         /*
           test to see if block ( time ) has an empty, if it is empty then threads must idle because their calculation would be useless.
           Otherwise, this will calculate the partial results of subtraction
           and squaring. Each element is stored in partial matrix which will
           be later summed and square rooted for the Euclidean distance. 
         */
         if( inputMat[ EmptyoffsetIndex ] != -1 ){
            //loop for thread stride
            while( tidx < cols*(bidx+1) )
               {  
                  //calc the column of the row that needs to be imputed
                  imputIdx = imputRow * cols + tidx - (bidx * cols);

                  
                  
                  //Calc difference between elements & square
                  diff = inputMat[imputIdx] - inputMat[tidx];
                  //print impute idx
                  printf("Impute index %d and tidx %d yield %f and %f \n", imputIdx, tidx, 
                                    inputMat[imputIdx], inputMat[tidx]);
                  printf("BID IS: %d \n", bidx);
                  
                  partialMat[tidx] = diff * diff;
                  //stride threads to next set of operations
                  tidx = tidx + blockDim.x;
               }
         //sync threads b4 reduction 
         __syncthreads();         
      
      //do reduction summation  
         //reset tidx from thread striding above
         tidx = bidx*cols + threadIdx.x + 2;  
         /*
           Calculate the index of element to be summed in reduction. 
           This will be a block size over to ensure no threads are summing
           element belonging to other thread. 
         */
         sumIdx = tidx + blockDim.x;
         /*
           stride loop for summing. The first block size number of
           threads will hold the sums. Then this will be reduced.
         */
         while( sumIdx < cols*(bidx+1) )
            {  
               /*
                 caclulate index of partial matrix that the reduction 
                 results are stored in, then sum and stride to next row
               */  
               partialMat[ tidx ] += partialMat[ sumIdx ];
               sumIdx+=blockDim.x;             
            }
            __syncthreads();  

      //thread reduction step
         //reset tidx
         tidx = bidx*cols + threadIdx.x + 2;      
         reduceThreads = blockDim.x / 2;
         while( reduceThreads > 0 )
            { 
               if( threadIdx.x < reduceThreads )
                  {
                     partialMat[ tidx ] += partialMat[ tidx + reduceThreads ];
                  }
               __syncthreads();   
               reduceThreads /= 2;   
            }            
            //Square root results of summation to get distance             
            partialMat[ (bidx * cols + 2) ] = 
                                   sqrt( partialMat[ (bidx * cols + 2) ] );
         }
        
         //stride to next set of blocks
         bidx+=gridDim.x;   
      }
}      

/*
  this function will transfer the second col of each row into an array so
  that sorting can be done on CPU
*/
__global__ void distXfer( float* inMat, float* outArr, int rows, int cols ){
   int bidx;
   bidx = blockIdx.x;
   //grid stride loop
   while( bidx < rows ){
      outArr[ bidx ] = inMat[ (bidx * cols + 2) ];
      bidx += gridDim.x;
   }
}     




