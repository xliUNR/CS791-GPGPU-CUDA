#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
////////////   CUTThrad Implementation  //////////////////////////////////////
/////////////////////   by Eric Li ////////////////////////////////////////////

#include <stdlib.h>
#include<stdio.h>


#include"cudafunctions.h"
#include "book.h"
 

/*
  declare struct that contains data ID, grid and block structure, as well as 3 pointers that will identify matrices that the kernel will work on. 
  a and b store input matrices
  c is for the results matrix 
*/  
struct dataStruct
   {
      int deviceID;
      int blocks;
      int * a;
      int * b;
      int * c;
      int * partial;
   };
/*
  This routine is called within the start_threads call. This will be run on all threads, each will call kernel on a seperate GPU.
*/
void* routine(void* dataSPtr)
   {
      dataStruct *data = (dataStruct*)dataSPtr;
      hipSetDevice(data->deviceID);
      //run kernel?
      helloThere<<<1,1>>>(data->deviceID, data->a, data->b, data->c);
      return 0;
   }

int main(int argc, char const *argv[])
{
   int numGPU, numGPUthreads;
   int N = 1;
   //get number of gpus
   hipGetDeviceCount(&numGPU);
   //initialize struct for data
   dataStruct *runData = new dataStruct[numGPU];
   //initialize thread array, each thread can be accessed by index
   CUTThread *thread = new CUTThread[numGPU];
   //CUTThread threadId[ MAX_GPU_COUNT];


   std::cout<< "Please enter in matrix dimensions: ";
   std::cin >> N;
   //calculate padding for reduction, needs to be power of 2
   numGPUthreads = N;
   //if odd, add 1 b
   if( numGPUthreads % 2 != 0 ){
     numGPUthreads+=1;
     }
   //check for power of 2, add 2 until it is power of 2
   while( ceil(log2((float)numGPUthreads-2)) 
                                    != floor(log2((float)numGPUthreads-2)) ){
     numGPUthreads+=2;
   }  


   //allocate unified memory and initialize beginning data
   for(int i=0; i < numGPU; i++){
      HANDLE_ERROR( hipMallocManaged(&(runData[i].a), N*N*sizeof(int)) );
      HANDLE_ERROR( hipMallocManaged(&(runData[i].b), N*N*sizeof(int)) );
      HANDLE_ERROR( hipMallocManaged(&(runData[i].c), N*N*sizeof(int)) );
      HANDLE_ERROR( hipMallocManaged(&(runData[i].partial), 
                                         N*N*numGPUthreads*sizeof(int)) );

      //fill array with data including 0 for result matrix
      for( int j=0; j < N*N; j++){
         runData[i].a[j] = 1;
         runData[i].b[j] = 1;
         runData[i].c[j] = 0;
      }
      //fill partial matrix with zeros
      for(int k=0; k < N*N*numGPUthreads; k++){
         runData[i].partial[k] = 0;
      }
      runData[i].deviceID = i;
   }

   //start threads
   for( int i = 0; i < numGPU; i++){
      thread[ i ] = start_thread(routine, &runData[i]);
   }

   //end threads
   /*for(int i=0; i < numGPU; i++){
      //end_thread( thread[i]);
      wait_for_threads(thread[i], NULL);
   }*/

   //end threads
   for(int i=0; i < numGPU; i++){
      end_thread( thread[i]);
      
   }

   //end threads
   for(int i=0; i < numGPU; i++){
      destroy_thread( thread[i]);
   }
   //print results
   for(int i=0; i< numGPU; i++){
      printf("\n Result from GPU: %d is %d", i, runData[i].c[0]);
   }


   //free memory
   for(int i=0; i<numGPU; i++){
      hipFree( runData[i].a );
      hipFree( runData[i].b );
      hipFree( runData[i].c );
      hipFree( runData[i].partial);
   }
   /* code */
   return 0;
}

//sequential implementation
void seqMatrixMult(int* in1, int* in2, int* output, int arrDim){
   //loop over column and rows for each element of the output matrix
   for(int i = 0; i < arrDim; i++){
      for(int j = 0; j < arrDim; j++){
         //initialize value of 0 for output matrix element
         output[ i*arrDim + j ] = 0;
         for(int k = 0; k < arrDim; k++){
            output[ i*arrDim + k ]+= in1[ i*arrDim + k ] * in2[ k*arrDim + j ];
         }
      }
   }

}