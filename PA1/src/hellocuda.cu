/*
  This program demonstrates the basics of working with cuda. We use
  the GPU to add two arrays. We also introduce cuda's approach to
  error handling and timing using cuda Events.

  This is the main program. You should also look at the header add.h
  for the important declarations, and then look at add.cu to see how
  to define functions that execute on the GPU.
 */

#include <iostream>

#include "add.h"

int main() {
  
  // Arrays on the host (CPU)
  int a[N][N], b[N][N], c[N][N];
  
  /*
    These will point to memory on the GPU - notice the correspondence
    between these pointers and the arrays declared above.
   */
  int *dev_a, *dev_b, *dev_c;

  /*
    These calls allocate memory on the GPU (also called the
    device). This is similar to C's malloc, except that instead of
    directly returning a pointer to the allocated memory, hipMalloc
    returns the pointer through its first argument, which must be a
    void**. The second argument is the number of bytes we want to
    allocate.

    NB: the return value of hipMalloc (like most cuda functions) is
    an error code. Strictly speaking, we should check this value and
    perform error handling if anything went wrong. We do this for the
    first call to hipMalloc so you can see what it looks like, but
    for all other function calls we just point out that you should do
    error checking.

    Actually, a good idea would be to wrap this error checking in a
    function or macro, which is what the Cuda By Example book does.
   */
  hipError_t err = hipMalloc( (void**) &dev_a, N * N * sizeof(int));
  if (err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(1);
  }
  hipMalloc( (void**) &dev_b, N * N * sizeof(int));
  hipMalloc( (void**) &dev_c, N * N * sizeof(int));

  // These lines just fill the host arrays with some data so we can do
  // something interesting. Well, so we can add two arrays.
  for (int i = 0; i < N; ++i) {

    for(int j=0; j < N; ++j){

      a[i][j] = 1;
      b[i][j] = 2;

    }
    
  }

 /*
    The following code is responsible for handling timing for code
    that executes on the GPU. The cuda approach to this problem uses
    events. For timing purposes, an event is essentially a point in
    time. We create events for the beginning and end points of the
    process we want to time. When we want to start timing, we call
    hipEventRecord.

    In this case, we want to record the time it takes to transfer data
    to the GPU, perform some computations, and transfer data back.
  */
  /*hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  hipEventRecord( start, 0 );*/

  /*
    Once we have host arrays containing data and we have allocated
    memory on the GPU, we have to transfer data from the host to the
    device. Again, notice the similarity to C's memcpy function.

    The first argument is the destination of the copy - in this case a
    pointer to memory allocated on the device. The second argument is
    the source of the copy. The third argument is the number of bytes
    we want to copy. The last argument is a constant that tells
    hipMemcpy the direction of the transfer.
   */
  hipMemcpy(dev_a, a, N * N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N * N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_c, c, N * N * sizeof(int), hipMemcpyHostToDevice);
  
  /*
    FINALLY we get to run some code on the GPU. At this point, if you
    haven't looked at add.cu (in this folder), you should. The
    comments in that file explain what the add function does, so here
    let's focus on how add is being called. The first thing to notice
    is the <<<...>>>, which you should recognize as _not_ being
    standard C. This syntactic extension tells nvidia's cuda compiler
    how to parallelize the execution of the function. We'll get into
    details as the course progresses, but for we'll say that <<<N,
    1>>> is creating N _blocks_ of 1 _thread_ each. Each of these
    threads is executing add with a different data element (details of
    the indexing are in add.cu). 

    In larger programs, you will typically have many more blocks, and
    each block will have many threads. Each thread will handle a
    different piece of data, and many threads can execute at the same
    time. This is how cuda can get such large speedups.
   */
  add<<<N, N>>>(dev_a, dev_b, dev_c);

  /*
    Unfortunately, the GPU is to some extent a black box. In order to
    print the results of our call to add, we have to transfer the data
    back to the host. We do that with a call to hipMemcpy, which is
    just like the hipMemcpy calls above, except that the direction of
    the transfer (given by the last argument) is reversed. In a real
    program we would want to check the error code returned by this
    function.
  */
  hipError_t err1 = hipMemcpy(c, dev_c, N * N * sizeof(int), hipMemcpyDeviceToHost);
  if (err1 != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(1);
  }


  /*
    This is the other end of the timing process. We record an event,
    synchronize on it, and then figure out the difference in time
    between the start and the stop.

    We have to call hipEventSynchronize before we can safely _read_
    the value of the stop event. This is because the GPU may not have
    actually written to the event until all other work has finished.
   */
  /*hipEventRecord( end, 0 );
  hipEventSynchronize( end );

  float elapsedTime;
  hipEventElapsedTime( &elapsedTime, start, end );*/

  /*
    Let's check that the results are what we expect.
   */
  /*for (int i = 0; i < N; ++i) {
    for(int j = 0; j < N; ++j){

      if (c[i][j] != a[i][j] + b[i][j]) {
      std::cerr << "Oh no! Something went wrong. You should check your cuda install and your GPU. :(" << std::endl;

      // clean up events - we should check for error codes here.
      hipEventDestroy( start );
      hipEventDestroy( end );

      // clean up device pointers - just like free in C. We don't have
      // to check error codes for this one.
      hipFree(dev_a);
      hipFree(dev_b);
      hipFree(dev_c);
      exit(1);
    }
    } 
    
  }*/

  /*
    Let's let the user know that everything is ok and then display
    some information about the times we recorded above.
   */
  std::cout << "Yay! Your program's results are correct." << std::endl;
  //std::cout << "Your program took: " << elapsedTime << " ms." << std::endl;
  
  // Cleanup in the event of success.
  hipEventDestroy( start );
  hipEventDestroy( end );

  /*hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);*/

}
