#include "hip/hip_runtime.h"

#include "add.h"

/*
  This is the function that each thread will execute on the GPU. The
  fact that it executes on the device is indicated by the __global__
  modifier in front of the return type of the function. After that,
  the signature of the function isn't special - in particular, the
  pointers we pass in should point to memory on the device, but this
  is not indicated by the function's signature.
 */
__global__ void add(int n, int *a, int *b, int *c) {

  /*
    Each thread knows its identity in the system. This identity is
    made available in code via indices blockIdx and threadIdx. This 
    equation calculates the unique ID for each element in the matrix
    since the memory is stored as a 1D list.
   */
  
  //0 for 1D grid of 1D blocks
  //1 for 1D grid of 2D blocks
  //2 for 2D grid of 1D blocks
  int option = 2;
  int thread_id;
    switch (option ) {
    case 0: 
       thread_id = threadIdx.x + blockIdx.x * blockDim.x;
       break;

    case 1:
       thread_id =  blockIdx.x * blockDim.x * blockDim.y
                          + threadIdx.y * blockDim.x + threadIdx.x;
       break;
       
    case 2:
       int blockId = blockIdx.y * gridDim.x + blockIdx.x;  
       thread_id = blockId * blockDim.x + threadIdx.x;
       break;                     

  }
  //int col = threadIdx.x + blockDim.x * blockIdx.x;
  //int row = threadIdx.y + blockDim.y * blockIdx.y;
  //int index = row * N + col;
  
  /*
    We make sure that the thread_id isn't too large, and then we
    assign c = a + b using the index we calculated above.

    The big picture is that each thread is responsible for adding one
    element from a and one element from b. Each thread is able to run
    in parallel, so we get speedup.
   */
   
  if (thread_id < n * n) {

    //c[thread_id] = a[thread_id] + b[thread_id];
    *(c + thread_id) = *(a + thread_id ) + *(b + thread_id);
    }
  /*if (col < N && row < N ) {

    //c[thread_id] = a[thread_id] + b[thread_id];
    c[index] = a[index] + b[index];
  }*/
}

//matrix add function that uses grid-striding
__global__ void strideAdd(int n, int *a, int *b, int *c) {
  //initialize offset AKA unique thread id
   int option = 2;
   int thread_id;
   switch (option ) {
    case 0: 
       thread_id = threadIdx.x + blockIdx.x * blockDim.x;
       break;

    case 1:
       thread_id =  blockIdx.x * blockDim.x * blockDim.y
                          + threadIdx.y * blockDim.x + threadIdx.x;
       break;
       
    case 2:
       int blockId = blockIdx.y * gridDim.x + blockIdx.x;  
       thread_id = blockId * blockDim.x + threadIdx.x;
       break;                     

  }

  //loop over each grid
  for( int i = thread_id; i < n*n; i+= blockDim.x * gridDim.x )
    {
      *(c + thread_id) = *(a + thread_id ) + *(b + thread_id);      
    }

}

/*//function to populate a matrix
__global__ void mat_init( int N, int *emptyMatrix ) {

  int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

  //check for valid memory location, then initialize element to 0
  if( thread_id < N * N )
  {
    //commented out one is for array of pointers
    //*((*(emptyMatrix)) + (blockId.x * blockDim.x + threadIdx.x)) = 0;
    *(emptyMatrix + (blockIdx.x * blockDim.x + threadIdx.x)) = 0;
  }
}*/


