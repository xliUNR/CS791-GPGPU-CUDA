#include "hip/hip_runtime.h"
#include <iostream>


#define N 2

//CUDA error handler provided by text
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void matpop(int N, int* emptyMatrix );







///////////////////// main function /////////////////////////////////////

int main(int argc, char const *argv[])
{
   int *dev_a;
   int host_a[N][N];
   //allocate memory


   HANDLE_ERROR( hipMalloc( (void**) &dev_a, N * N * sizeof(int)));

   //initialize matrix on device using parallel and copy over
   matpop<<<N,N>>>(N, dev_a);
   hipMemcpy(host_a, dev_a, N * N * sizeof(int), hipMemcpyDeviceToHost);

   printf( "\n Matrix: \n");
   //prints matrix elements
    for (int i = 0; i < N; i++){

          for (int j = 0; j < N; j++){
	  
	      printf ("%i ", *((host_a) + (i * N + j)));

	      }

      printf ("\n");
    }
   
	return 0;
}




//////////////////// function declarations ////////////////////////////



//populates a matrix on device
__global__ void matpop( int N, int* emptyMatrix ){
	
	int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

	//check for valid memory location, then initialize element to 0
	if( thread_id < N * N )
	{
		//commented out one is for array of pointers
		//*((*(emptyMatrix)) + (blockId.x * blockDim.x + threadIdx.x)) = 0;
		*(emptyMatrix + (blockIdx.x * blockDim.x + threadIdx.x)) = 0;
	}
	
}

