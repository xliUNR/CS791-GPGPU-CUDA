#include "hip/hip_runtime.h"
#include <iostream>


#define N 2


__global__ voic matpop(int N, int* emptyMatrix );







///////////////////// main function /////////////////////////////////////

int main(int argc, char const *argv[])
{
   int *dev_a, *host_a;
   int host_a[N][N];
   //allocate memory


   HANDLE_ERROR( hipMalloc( (void**) &dev_a, N * N * sizeof(int)));

   //initialize matrix on device using parallel and copy over
   matpop<<<N,N>>>(N, dev_a);
   hipMemcpy(host_a, dev_a, N * N * sizeof(int), hipMemcpyDeviceToHost);

   printf( "\n Matrix: \n");
   //prints matrix elements
    for (int i = 0; i < N; i++){

          for (int j = 0; j < N; j++){
	  
	      printf ("%i ", *((host_a) + (i * N + j)));

	      }

      printf ("\n");
    }
   
	return 0;
}




//////////////////// function declarations ////////////////////////////

//CUDA error handler provided by text
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))\


//populates a matrix on device
__global__ void matpop( int N, int* emptyMatrix ){
	
	int thread_id = threadIdx.x + blockId.x * blockDim.x;

	//check for valid memory location, then initialize element to 0
	if( thread_id < N * N )
	{
		//commented out one is for array of pointers
		//*((*(emptyMatrix)) + (blockId.x * blockDim.x + threadIdx.x)) = 0;
		*((emptyMatrix) + (blockId.x * blockDim.x + threadIdx.x)) = 0;
	}
	
}

